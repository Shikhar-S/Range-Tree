#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include <algorithm>
#include <utility>
#include <hip/hip_runtime.h>
#include <cstdio>

#define HANDLE_ERROR(x) if((x)!=hipSuccess){std::cout<<hipGetErrorString((x))<<std::endl;exit(-1);}
#define MAX_DEPTH 16

struct Point{
	int x; int y;
	Point(int _x, int _y) : x(_x), y(_y) {}
};
struct Query{
	int a, b, c, d;
	Query(int _a, int _b, int _c, int _d) : a(_a), b(_b), c(_c), d(_d) {}
};
void swap(int &a, int &b)
{
	int t = a;
	a = b;
	b = t;
}
int N;
int logN;
int q;
__device__ void selection_sort_x(int left, int right, Point* output)
{
	for (int i = left; i <= right; i++)
	{
		int min_idx = i;
		for (int j = i + 1; j <= right; j++)
		{
			if (output[j].x < output[min_idx].x)
			{
				min_idx = j;
			}
		}
		if (i != min_idx){
			Point t = output[i];
			output[i] = output[min_idx];
			output[min_idx] = t;
		}

	}
}

__global__ void quicksort_x(int left, int right, Point *output, int depth)
{
	if (depth >= MAX_DEPTH)
	{
		selection_sort_x(left, right, output);
		return;
	}
	Point *lptr = output + left;
	Point *rptr = output + right;
	Point pivot = output[left + (right - left) / 2];

	while (lptr <= rptr)
	{
		Point lval = *lptr;
		Point rval = *rptr;
		while (lval.x < pivot.x)
		{
			lptr++;
			lval = *lptr;
		}
		while (rval.x > pivot.x)
		{
			rptr--;
			rval = *rptr;
		}
		if (lptr <= rptr)
		{
			*lptr++ = rval;
			*rptr-- = lval;
		}
	}

	int new_right = rptr - output;
	int new_left = lptr - output;

	if (left < (rptr - output))
	{
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
		quicksort_x <<<1, 1, 0, s >>>(left, new_right, output, depth + 1);
		hipStreamDestroy(s);
	}

	if ((lptr - output)<right)
	{
		hipStream_t s1;
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		quicksort_x <<<1, 1, 0, s1 >>>(new_left, right, output, depth + 1);
		hipStreamDestroy(s1);
	}
}
__global__ void merge_v2(Point* d_secondary,int canonical_size,int call_num,int N,int* x,int* y)
{
	int offset_first = (call_num)*N;
	int offset_block = canonical_size*blockIdx.x; //one block is responsible for one canonical sized node

	int num_of_threads = blockDim.x;
	int idx = threadIdx.x;
	bool flag = false;
	if (idx == 0)
	{
		x[idx + blockDim.x*blockIdx.x] = 0;
		y[idx + blockDim.x*blockIdx.x] = 0;
		flag = true;
	}
	
	int n = canonical_size / 2;
	int A_start = idx*(2 * n) / num_of_threads; //without offsets
	int B_start =  max(0, A_start - (n - 1));
	A_start =min(n - 1, A_start);
	int length_of_array;

	if (B_start == 0)
	{

		length_of_array = A_start + 1;
	}
	else
		length_of_array = n - B_start;

	int left = 0, right = length_of_array - 1;
	
	idx = blockDim.x*blockIdx.x + threadIdx.x;
	while (left <= right && !flag)
	{
		
		int mid = left + (right - left) / 2;
		int I =  A_start - mid;
		int i = offset_first + offset_block + I;
		int J =  B_start + mid;
		int j = offset_first + offset_block + n + J;
		if (d_secondary[i].y > d_secondary[j].y)
		{
			left = mid + 1;
		}
		else
		{
			if (J == 0)
			{
				x[idx] = (I + 1);
				y[idx] = (J);
				flag = true;
			}
			else if (I == n - 1)
			{
				x[idx] = (I + 1);
				y[idx] = (J);
				flag = true;
			}
			else
			{
				if (d_secondary[i + 1].y > d_secondary[j - 1].y)
				{
					x[idx] = (I + 1);
					y[idx] = (J);
					flag = true;
				}
				else
				{
					right = mid;
				}
			}
		}
	}
	left--;
	if (!flag)
	{
		x[idx] = (A_start - left);
		y[idx] = (n);
	}
	__syncthreads();

	int end_x, end_y;
	if (idx-(blockDim.x*blockIdx.x) == num_of_threads - 1)
	{
		end_x = n;
		end_y = n;
	}
	else
	{
		end_x = x[idx + 1];
		end_y = y[idx + 1];
	}
	int cur_x = x[idx];
	int cur_y = y[idx];
	int put_at = cur_x + cur_y;
	end_x += offset_first + offset_block;
	end_y += offset_first + offset_block + n;
	cur_x += offset_first + offset_block;
	cur_y += offset_first + offset_block + n;
	put_at += offset_first + offset_block + N;
	while (cur_x<end_x && cur_y<end_y)
	{
		if (d_secondary[cur_x].y <= d_secondary[cur_y].y)
		{
			d_secondary[put_at++] = d_secondary[cur_x++];
		}
		else
		{
			d_secondary[put_at++] = d_secondary[cur_y++];
		}
	}
	while (cur_x<end_x)
		d_secondary[put_at++] = d_secondary[cur_x++];
	while (cur_y<end_y)
		d_secondary[put_at++] = d_secondary[cur_y++];
}
void build_secondary_tree(Point *d_secondary, int canonical_size, int call_num)
{
	if (canonical_size >= 2 * N)
	{
		return;
	}
	//std::cout << "Calling with " << canonical_size << std::endl;
	/*include logic to find the correct number of threads and blocks to run*/
	int num_of_threads_per_block = canonical_size;
	if (num_of_threads_per_block > 1024)
		num_of_threads_per_block = 1024;
	int blocks = N / canonical_size;


	int *d_index_x;
	int *d_index_y;

	HANDLE_ERROR(hipMalloc((void**)&d_index_x,sizeof(int)*num_of_threads_per_block*blocks));
	HANDLE_ERROR(hipMalloc((void**)&d_index_y, sizeof(int)*num_of_threads_per_block*blocks));
	HANDLE_ERROR(hipDeviceSynchronize());

	merge_v2 <<< blocks, num_of_threads_per_block >> > (d_secondary, canonical_size, call_num, N, d_index_x, d_index_y);
	//merge << <1, (N / canonical_size) >> >(d_secondary, canonical_size, call_num, N);
	
	HANDLE_ERROR(hipFree(d_index_x));
	HANDLE_ERROR(hipFree(d_index_y));
	HANDLE_ERROR(hipDeviceSynchronize());
	build_secondary_tree(d_secondary, canonical_size * 2, call_num + 1);
}

void build_tree(Point *d_primary, Point *d_secondary, Point *points, Point *primary, Point *secondary)
{
	int left = 0;
	int right = N - 1;
	HANDLE_ERROR(hipMemcpy(d_primary, points, N*sizeof(Point), hipMemcpyHostToDevice));
	quicksort_x << <1, 1 >> >(left, right, d_primary, 0);
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipMemcpy(d_secondary, d_primary, N*sizeof(Point), hipMemcpyDeviceToDevice)); //building the first level
	build_secondary_tree(d_secondary, 2, 0);
	HANDLE_ERROR(hipMemcpy(primary, d_primary, N*sizeof(Point), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(secondary, d_secondary, N*logN*sizeof(Point), hipMemcpyDeviceToHost));
}
void print_tree(Point* primary, Point* secondary)
{
	std::cout << "-----------------------------------------------------Primary tree-------------------------------------------------------------------\n";
	for (int i = 0; i < N; i++)
	{
		std::cout << primary[i].x << " " << primary[i].y << std::endl;
	}
	std::cout << "-----------------------------------------------------Secondary tree-----------------------------------------------------------------\n";
	for (int i = 0; i < N*logN; i++)
	{
		if (i%N == 0)
		{
			std::cout << "____________________________________________________________________\n";
		}
		std::cout << secondary[i].x << " " << secondary[i].y << std::endl;
		
	}
}

int main()
{
	FILE *f = fopen("f.txt", "r");
	if (!f)
	{
		std::cout << "ERROR\n";
		exit(-1);
	}
	//std::cin >> N;
	fscanf(f, "%d", &N);
	logN = 0;
	int cur = 1;
	while (cur < N)
	{
		cur = cur * 2;
		logN++;
	}
	logN++;
	//std::cout << logN << std::endl;
	Point *input = (Point*)malloc(N*sizeof(Point));
	Point *primary = (Point*)malloc(N*sizeof(Point));
	Point *secondary = (Point*)malloc(N*logN*sizeof(Point));

	for (int i = 0; i < N; i++)
	{
		int x, y;
		//std::cin >> x >> y;
		fscanf(f, "%d %d", &x, &y);
		Point t(x, y);
		input[i] = t;
	}

	Point *d_primary;
	Point *d_secondary;

	HANDLE_ERROR(hipMalloc((void**)&d_primary, N*sizeof(Point)));
	HANDLE_ERROR(hipMalloc((void**)&d_secondary, N*logN*sizeof(Point)));

	build_tree(d_primary, d_secondary, input, primary, secondary);
	print_tree(primary, secondary);

	HANDLE_ERROR(hipFree(d_primary));
	HANDLE_ERROR(hipFree(d_secondary));
	free(primary);
	free(secondary);
	
	return 0;
}
